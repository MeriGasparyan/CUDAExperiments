#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <cmath> // for ceil if needed

__global__
void vecAddKernel(float* A, float* B, float* C, int n) {
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    if (i < n) {
        C[i] = A[i] + B[i];
    }
}

void vecAdd(float* A, float* B, float* C, int n) {
    float *A_d, *B_d, *C_d;
    size_t size = n * sizeof(float);

    hipMalloc((void**)&A_d, size);
    hipMalloc((void**)&B_d, size);
    hipMalloc((void**)&C_d, size);

    hipMemcpy(A_d, A, size, hipMemcpyHostToDevice);
    hipMemcpy(B_d, B, size, hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;

    vecAddKernel<<<blocksPerGrid, threadsPerBlock>>>(A_d, B_d, C_d, n);

    hipMemcpy(C, C_d, size, hipMemcpyDeviceToHost);

    hipFree(A_d);
    hipFree(B_d);
    hipFree(C_d);
}


int main() {
    int n = 1 << 10; // 1024 elements
    float *A = new float[n];
    float *B = new float[n];
    float *C = new float[n];

    for (int i = 0; i < n; i++) {
        A[i] = 1.0f;
        B[i] = 2.0f;
    }

    vecAdd(A, B, C, n);

    // Print first 5 results
    for (int i = 0; i < 5; i++) {
        std::cout << A[i] << " + " << B[i] << " = " << C[i] << std::endl;
    }

    delete[] A;
    delete[] B;
    delete[] C;
    return 0;
}